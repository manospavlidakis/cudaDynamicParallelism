#include "hip/hip_runtime.h"
#include <builtin_types.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#include <stdlib.h>
#include <sys/time.h>
#include <iostream>
#include <unistd.h>
#include "../drvapi_error_string.h"

#define iterations 1000000
#define POLL
#define TRAP
using namespace std;

//__device__ __managed__ volatile int kill = 0;
__global__ void another_child(float *out, int n ) 
{
	//	printf("inside child!!\n");
	for (int i = 0; i < n; i++)
	{
		out[i] = 88888; 
	}	
}
__global__ void child_launch(float *out, int n, int *kill ) 
{
	//	printf("inside child!!!\n");
	//for (int iter=0; iter<iterations; iter++)
	//{
		for (int i = 0; i < n; i++)
		{
			out[i] = 12; 
		}
	//}
	//in case that there is no need to kill
	//wait for completion and then make kile
	//flag tn 1 in order to exit from parent
	//also!!
	*kill = 1;
}

__global__ void parent_launch(float *out, int n, int *kill) 
{
	if (threadIdx.x == 0) 
	{
		child_launch <<<512, 512>>>(out, n, kill);
		//	printf("parent poll %d: \n",poll);

		while (1)
		{
			if (*kill == 1)
			{
#ifdef TRAP

				printf("TRAP %d: \n",*kill);
				asm("trap;"); 
#else 
				printf("EXIT %d: \n",*kill);
				asm("exit;");
#endif
			}
		}
	}
	*kill=0;
}


void kernel_before_kill(int *kill)
{
	int N=100;
	hipError_t err;
	struct timeval kill_st, kill_end;
	float *d_out;
	float h_in[N], h_out[N];

	//float *h_in, *h_out;
	//h_in = (float*)malloc(N*sizeof(float));
	//h_out = (float*)malloc(N*sizeof(float));
	for (int i = 0; i<N; i++)
	{
		h_in[i] = i;
		h_out[i] = 0;
	}

	//Start timer
	gettimeofday(&kill_st,NULL);
	double t1 = kill_st.tv_sec  * 1000000 +  kill_st.tv_usec;
	{
		err = hipMalloc((void **)&d_out, N * sizeof(float));
		if (err != hipSuccess) cerr<<"Malloc: "<<hipGetErrorString(err)<<endl;

		err = hipMemcpy(d_out, h_in, N * sizeof(float), hipMemcpyHostToDevice);
		if (err != hipSuccess) cerr<<"Memcpy (H2D): "<<hipGetErrorString(err)<<endl;
		child_launch<<< 1, 1 >>>(d_out, N*sizeof(float), kill);
		err=hipGetLastError();
		if (err != hipSuccess) cerr<<"Krnl: "<<hipGetErrorString(err)<<endl;

		err = hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);
		if (err != hipSuccess) cerr<<"Mecpy (D2H): "<<hipGetErrorString(err)<<endl;

		err = hipFree(d_out);
		if (err != hipSuccess) cerr<<"Free: "<<hipGetErrorString(err)<<endl;
	}
	gettimeofday(&kill_end,NULL);
	double t2 = kill_end.tv_sec  * 1000000 +  kill_end.tv_usec;
	long double duration = (t2 - t1)/1000 ;
	cout<<"Duration of kernel (without kill): "<<duration<<" ms"<<endl;

	//free(h_in);
	//free(h_out);
}

void kernel_killed(int *kill)
{
	cerr<<"Kernel kill!!!"<<endl;
	int N=100;
	hipError_t err;
	struct timeval kill_st, kill_end;
	float *d_out;
	float *h_in, *h_out;

	h_in = (float*)malloc(N*sizeof(float));
	h_out = (float*)malloc(N*sizeof(float));
	for (int i = 0; i<N; i++)
	{
		h_in[i] = i;
		h_out[i] = 0;
	}

	//Start timer
	gettimeofday(&kill_st,NULL);
	double t1 = kill_st.tv_sec  * 1000000 +  kill_st.tv_usec;
	{

		//hipMallocManaged(&kill, sizeof(int));
		err = hipMalloc((void **)&d_out, N * sizeof(float));
		if (err != hipSuccess) cerr<<"Malloc: "<<hipGetErrorString(err)<<endl;

		err = hipMemcpy(d_out, h_in, N * sizeof(float), hipMemcpyHostToDevice);
		if (err != hipSuccess) cerr<<"Memcpy (H2D): "<<hipGetErrorString(err)<<endl;


		//set kill (kernel is going to be killed)
		parent_launch<<< 1, 1 >>>(d_out, sizeof(float), kill);
		usleep(100);
		*kill = 1;
		err=hipDeviceSynchronize();
		if (err != hipSuccess) cerr<<"Krnl: "<<hipGetErrorString(err)<<endl;
/*
		err = hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);
		if (err != hipSuccess) cerr<<"Mecpy (D2H): "<<hipGetErrorString(err)<<endl;
*/
//		err = hipFree(d_out);
//		if (err != hipSuccess) cerr<<__LINE__<<" Free: "<<hipGetErrorString(err)<<endl;
		//unset kill (next kernel is not going to be killed)
//		*kill = 0;

	}
	gettimeofday(&kill_end,NULL);
	double t2 = kill_end.tv_sec  * 1000000 +  kill_end.tv_usec;
	long double duration = (t2 - t1)/1000 ;
	cout<<"Duration of kernel (killed): "<<duration<<" ms"<<endl;

	free(h_in);
	free(h_out);
}

void kernel_after_kill(int *kill)
{
	int N=100;
	hipError_t err;
	struct timeval kill_st, kill_end;
	float *dev_out;
	float *host_in, *host_out;

	host_in = (float*)malloc(N*sizeof(float));
	host_out = (float*)malloc(N*sizeof(float));
	for (int i = 0; i<N; i++)
	{
		host_in[i] = i;
		host_out[i] = 0;
	}

	//Start timer
	gettimeofday(&kill_st,NULL);
	double t1 = kill_st.tv_sec  * 1000000 +  kill_st.tv_usec;
	{
		err = hipMalloc((void **)&dev_out, N * sizeof(float));
		if (err != hipSuccess) cerr<<"Malloc: "<<hipGetErrorString(err)<<endl;

		err = hipMemcpy(dev_out, host_in, N * sizeof(float), hipMemcpyHostToDevice);
		if (err != hipSuccess) cerr<<"Memcpy (H2D): "<<hipGetErrorString(err)<<endl;
		child_launch<<< 1, 1 >>>(dev_out, N*sizeof(float), kill);
		err=hipGetLastError();
		if (err != hipSuccess) cerr<<"Krnl: "<<hipGetErrorString(err)<<endl;

		err = hipMemcpy(host_out, dev_out, N * sizeof(float), hipMemcpyDeviceToHost);
		if (err != hipSuccess) cerr<<"Mecpy (D2H): "<<hipGetErrorString(err)<<endl;

		err = hipFree(dev_out);
		if (err != hipSuccess) cerr<<"Free: "<<hipGetErrorString(err)<<endl;
	}
	gettimeofday(&kill_end,NULL);
	double t2 = kill_end.tv_sec  * 1000000 +  kill_end.tv_usec;
	long double duration = (t2 - t1)/1000 ;
	cout<<"Duration of kernel (after kill): "<<duration<<" ms"<<endl;	

	free(host_in);
	free(host_out);
}


int main (int argc, char * argv[])
{
	int N=100;
	struct timeval kill_st, kill_end;
	float *d_out;
	float *h_in, *h_out;


	hipError_t result;
	hipDevice_t cuDevice;

	
	int *kill = 0;
	hipError_t err;
	err = hipSetDevice(0);	
	if (err != hipSuccess)
	{
		cerr<<"failed to se device"<<endl;
	}
	//Get device of the current thread
	result = hipDeviceGet (&cuDevice, 1);
	if (result != hipSuccess)
	{
		printf("hipDeviceGet: %d\n", getCudaDrvErrorString(result));
		return 0;
	}
	hipMallocManaged((void**)&kill, sizeof(int));

/*	cerr<<"->>>>>>>> kernel_before_kill: "<<*kill<<endl;
	kernel_before_kill(kill);
	err = hipGetLastError();
	if (err != hipSuccess) cerr<<"kernel_before_kill failed: "<<hipGetErrorString(err)<<endl;
	cerr<<"============================================="<<endl;
	sleep(1);	
*/
	cerr<<"->>>>>>>> kernel_killed: "<<*kill<<endl;
	kernel_killed(kill);
	//err = hipGetLastError();
	//if (err != hipSuccess) cerr<<"kernel_killed: "<<hipGetErrorString(err)<<endl;

	cerr<<"============================================="<<endl;
	cerr<<"Done !!! Press Ctrl+c to exit..."<<endl;
	while(1);

}
