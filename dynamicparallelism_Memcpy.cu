#include <builtin_types.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#include <stdlib.h>
#include <sys/time.h>
#include <iostream>
#include <unistd.h>
using namespace std;

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	   if (code != hipSuccess) 
		      {
			            fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
				          if (abort) exit(code);
					     }
}

//__device__ __managed__ int poll=0;

__global__ void child_launch(float *in, float *out, int n ) 
{
	while(1){
//		printf("inside child!!\n");
		for (int i = 0; i < n; i++)
		{
			out[i] = 12; 
		}	
	}
}

//__global__ void parent_launch(float *in, float *out, int n, int *poll ) 
__global__ void parent_launch(float *in, float *out, int n, int *devPoll) 
{
	if (threadIdx.x == 0) 
	{
		child_launch<<< 16, 1 >>>(in, out, n);
		//cudaDeviceSynchronize();
	}

	while (1)
	{
//		printf("%d\n", *devPoll);
		if (*devPoll == 1)
			asm("trap;"); 
	}

}


int main ()
{
	struct timeval kill_st, kill_end;
	int N = 10;
	float *d_in, *d_out, *h_in, *h_out;
	int *poll;
	int *devPoll;
	hipError_t err;
	err = hipSetDevice(3);	
	if (err != hipSuccess)
	{
		cerr<<"failed to se device"<<endl;
	}

	h_in = (float*)malloc(N*sizeof(float));
	h_out = (float*)malloc(N*sizeof(float));

	for (int i = 0; i<N; i++)
	{
		h_in[i] = i;
		h_out[i] = 0;	
	}
	cudaErrorCheck(hipMalloc(&d_in, N * sizeof(float)));
	cudaErrorCheck(hipMalloc(&d_out, N * sizeof(float)));
	cudaErrorCheck(hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice));

	cudaErrorCheck(hipHostMalloc((void**)&poll, sizeof(int), hipHostMallocDefault));
	cudaErrorCheck(hipMalloc((void**)&devPoll, sizeof(int)));
	//cudaErrorCheck(cudaMemset((void**)&devPoll, 0, sizeof(int)));
	
	cout<<"start parent kernel"<<endl;

	gettimeofday(&kill_st,NULL);
	double t1 = kill_st.tv_sec  * 1000000 +  kill_st.tv_usec;

	//parent_launch<<< 1, 1 >>>(d_in, d_out, sizeof(float), poll);
	parent_launch<<< 1, 1 >>>(d_in, d_out, sizeof(float), devPoll);
	//sleep(1);		

	cout<<"Poll == "<<*poll<<endl;
	*poll = 1;
	cout<<"Poll == "<<*poll<<endl;
	
	cudaErrorCheck(hipMemcpy(devPoll, &poll, sizeof(int), hipMemcpyHostToDevice));

//	cudaDeviceSynchronize();	
	gettimeofday(&kill_end,NULL);
	double t2 = kill_end.tv_sec  * 1000000 +  kill_end.tv_usec;

	long double duration = (t2 - t1)/1000 ;
	cout<<"Duration of kernel ( "<< poll<< "): "<<duration<<" ms"<<endl;

	//	cout<<"2. Poll : "<<poll<<endl;
	cudaErrorCheck(hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost));
	/*	
		for (int i = 0; i< N ;i++)
		cout <<" OUT: "<<h_out[i]<<" ,";
	 */
	cout<<endl;
	cerr<<"Done !!!"<<endl;
//	while(1);
}
